#include "CudaEventTimer.h"
#include <stdexcept>

CudaEventTimer::CudaEventTimer() {
    // Create start and stop events
    hipError_t err = hipEventCreate(&startEvent);
    if (err != hipSuccess) {
        throw std::runtime_error("Failed to create start event");
    }
    err = hipEventCreate(&stopEvent);
    if (err != hipSuccess) {
        hipEventDestroy(startEvent);
        throw std::runtime_error("Failed to create stop event");
    }
}

CudaEventTimer::~CudaEventTimer() {
    // Destroy events
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
}

void CudaEventTimer::start(hipStream_t stream) {
    // Record start event
    hipEventRecord(startEvent, stream);
}

void CudaEventTimer::stop(hipStream_t stream) {
    // Record stop event
    hipEventRecord(stopEvent, stream);
}

void CudaEventTimer::synchronize() {
    // Synchronize stop event
    hipEventSynchronize(stopEvent);
}

bool CudaEventTimer::query() {
    // Query stop event
    hipError_t err = hipEventQuery(stopEvent);
    return err == hipSuccess;
}

float CudaEventTimer::getElapsedTime() {
    // Get elapsed time between start and stop events
    float ms = 0.0f;
    hipEventElapsedTime(&ms, startEvent, stopEvent);
    return ms;
}
