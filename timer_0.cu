//time code using CUDA events

hipEvent_t start, stop;
float time;

hipEventCreate(&start);
hipEventCreate(&stop);

hipEventRecord( start, 0 );
kernel<<<grid,threads>>> ( d_odata, d_idata, size_x, size_y,
                           NUM_REPS);
hipEventRecord( stop, 0 );
hipEventSynchronize( stop );

hipEventElapsedTime( &time, start, stop );
hipEventDestroy( start );
hipEventDestroy( stop );
